#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <string>
#include <thrust/device_vector.h>
#include <fstream>
#include <time.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdio.h>

#define PROFILE_ALL_EVENTS_METRICS 0
int counter1 = 1;

int numARows = 64;
int numACols = 64;
int numBCols = 64;

__global__ void matMul(float* A, float* B, float* C, int numARows, int numACols, int numBCols) {
    // compute global thread coordinates
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // linearize coordinates for data access
    int offset = row * numBCols + col;

    if ((row < numARows) && (col < numBCols)) {
        float cumSum = 0;
        for (int k = 0; k < numACols; k++) {
            cumSum += A[row*numACols + k] * B[k*numBCols + col];
        }
        C[offset] = cumSum;
    }
}




static void compute() {

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    size_t sizeA = numARows * numACols * sizeof(float);
    size_t sizeB = numACols * numBCols * sizeof(float);
    size_t sizeC = numARows * numBCols * sizeof(float);

    // allocate host memory
    float* h_A = (float*)malloc(sizeA);
    float* h_B = (float*)malloc(sizeB);
    float* h_C = (float*)malloc(sizeC);

    // initialize host matrices
    int i, j, offset;
    for (i = 0; i <  numARows; i++) {
        for (j = 0; j < numACols; j++) {
            offset = i*numACols + j;
            h_A[offset] = i;
        }
    }
    for (i = 0; i <  numACols; i++) {
        for (j = 0; j < numBCols; j++) {
            offset = i*numBCols + j;
            h_B[offset] = i;
        }
    }

    // allocate device matrices
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, sizeA);
    hipMalloc((void**)&d_B, sizeB);
    hipMalloc((void**)&d_C, sizeC);

    // transfer to GPU
    hipMemcpy(d_A, h_A, sizeA, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeB, hipMemcpyHostToDevice);

    // kernel launch
    

    // dim3 threadPerBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    // dim3 blockPerGrid(ceil(numBCols/(float)BLOCK_SIZE), ceil(numACols/(float)BLOCK_SIZE), 1);


    hipEventRecord(start);
    for(int i = 0 ; i<10; i++){
    matMul<<<16,128>>>(d_A, d_B, d_C, numARows, numACols, numBCols);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);


    hipMemcpy(h_C, d_C, sizeC, hipMemcpyDeviceToHost);


    free(h_A); free(h_B); free(h_C); 
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

}

 int main()  
 {
  
 
   compute();
 
 
 
   
 }
 
